#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n) 
{
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) 
    {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        if(idx >= n*n)
        {
            // n may not divisible by BLOCK_SIZE
            tile_a[threadIdx.y][threadIdx.x] = 0;
        }
        else
        {
            tile_a[threadIdx.y][threadIdx.x] = d_a[idx];
        }

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        if(idx >= n*n)
        {
            tile_b[threadIdx.y][threadIdx.x] = 0;
        }  
        else
        {
            tile_b[threadIdx.y][threadIdx.x] = d_b[idx];
        }
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) 
        {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }
    /*
    if(row < n && col < n)
    {
        d_result[row * n + col] = tmp;
    }
    */
}


int main(int argc, char const *argv[])
{
    int m = 4096;
    int n = 4096;
    int k = 4096;
    srand(3333);

    // allocate memory in host RAM, h_cc is used to store CPU result
      
    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k);
    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k);

    // random initialize matrix A
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < n; ++j) {
            h_a[i * n + j] = rand() % 1024;
        }
    }

    // random initialize matrix B
    for (int i = 0; i < n; ++i) {
        for (int j = 0; j < k; ++j) {
            h_b[i * k + j] = rand() % 1024;
        }
    }

    // copy matrix A and B from host to device memory
    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);
    
    // Launch kernel 
    for(int i=0; i<10; i++){
        gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);    
    }
    hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);

    
    return 0;
}